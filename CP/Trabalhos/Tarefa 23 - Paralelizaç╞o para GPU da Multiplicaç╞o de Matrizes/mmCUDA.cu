
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/**
 * ==========================================================
 *            CÓDIGOS TESTADOS NO SERVIDOR PARCODE             
 * ==========================================================
 * Tempo código sequencial    = 1m6.872s 
 * Tempo código paralelo      = 0m17.669s ---- SpeedUp: 03,78
 * Tempo código paralelo GPU  = 0m5.449s  ---- SpeedUp: 12,27
 * Tempo código paralelo CUDA = 0m2.812s  ---- SpeedUp: 23,78
 * ==========================================================
 *      WARPS PARA MELHOR VERSÃO DE PARALELIZÇÃO NA GPU       
 * ==========================================================
 * Event Name                 Min     Max     Avg     Total
 * warps_launched              72      72      72        72
 * ==========================================================
 * Metric Name                     Min       Max       Avg
 * warp_execution_efficiency      86,81%    86,81%    86,81%
 * ==========================================================
 */

__global__ void mm_cuda(double* a, double* b, double* c, int width) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;

    if (i < width) {
        if (j < width) {
            double sum = 0;
            for (int k = 0; k < width; k++) {
                double x = a[i * width + k];
                double y = b[k * width + j];
                sum += x * y;
            }
            c[i * width + j] = sum;
        }
    }
}

int main()
{
    int width = 2000;
    double *a = (double*) malloc (width * width * sizeof(double));
    double *b = (double*) malloc (width * width * sizeof(double));
    double *c = (double*) malloc (width * width * sizeof(double));

    for(int i = 0; i < width; i++) {
        for(int j = 0; j < width; j++) {
            a[i*width+j] = i;
            b[i*width+j] = j;
            c[i*width+j] = 0;
        }
    }

  int size = (width * width)*sizeof(double);
  double *d_a, *d_b, *d_c;

  hipMalloc((void **) &d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_b, size);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_c, size);

  int block_size = 32;
  dim3 dimGrid((width-1)/block_size + 1, (width-1)/block_size + 1, 1);
  dim3 dimBlock(block_size,block_size,1);

  mm_cuda<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, width);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

}